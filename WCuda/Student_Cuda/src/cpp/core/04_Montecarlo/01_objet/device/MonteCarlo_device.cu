#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>



__global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbDarts,uint m, uint* ptrDevNxTotal);


__device__ float f(float x);


__global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbDarts, uint m, uint* ptrDevNxTotal)
    {
    extern __shared__ uint TAB_SM[];

    const int NB_THREAD=Indice1D::nbThread();
    const int TID=Indice1D::tid();
    const int TIDLocal = Indice1D::tidLocal();

    hiprandState generator = tabDevGeneratorGM[TID];

    int localNx = 0;
    float x;
    float y;

    for(int i = 0; i < nbDarts; i++)
	{
	x = hiprand_uniform(&generator);
	y = hiprand_uniform(&generator) * m;

	if ( y  < f(x) )
	    {
	    localNx ++;
	    }
	}

    TAB_SM[TIDLocal] = localNx;

    __syncthreads();

    reductionADD<uint>(TAB_SM, ptrDevNxTotal);
    }




__device__ float f(float x)
    {
    return 4/(1+x*x);
    }
